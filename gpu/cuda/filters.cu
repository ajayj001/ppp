#include "hip/hip_runtime.h"

#include <Timer.hpp>
#include <iostream>
#include <iomanip>
#include "defines.h"

using LOFAR::NSTimer;
using std::cout;
using std::cerr;
using std::endl;
using std::fixed;
using std::setprecision;


void checkError(hipError_t error, const char* description) {
	if (error != hipSuccess) {
		fprintf(stderr, description, hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
}

__global__ void
rgb2gray_kernel(uchar *inputImage, uchar *grayImage, const int width, const int height, const size_t pitch) {
	const int x = blockDim.x * blockIdx.x + threadIdx.x;
	const int y = blockDim.y * blockIdx.y + threadIdx.y;

	// Make sure we are within bounds
	if (x >= width || y >= height) return;

	// Fetch 3 times 4 pixels from device buffer
	const int quarterPitch = pitch / 4;
	const uchar4 red = ((uchar4*)inputImage)[(y * quarterPitch) + x];
	const uchar4 green = ((uchar4*)inputImage)[(quarterPitch * height) + (y * quarterPitch) + x];
	const uchar4 blue = ((uchar4*)inputImage)[(2 * quarterPitch * height) + (y * quarterPitch) + x];

	// Calculate grey values for 4 pixels
	uchar4 grey;
	grey.x = (RED_COEFFICIENT * red.x) + (GREEN_COEFFICIENT * green.x) + (BLUE_COEFFICIENT * blue.x);
	grey.y = (RED_COEFFICIENT * red.y) + (GREEN_COEFFICIENT * green.y) + (BLUE_COEFFICIENT * blue.y);
	grey.z = (RED_COEFFICIENT * red.z) + (GREEN_COEFFICIENT * green.z) + (BLUE_COEFFICIENT * blue.z);
	grey.w = (RED_COEFFICIENT * red.w) + (GREEN_COEFFICIENT * green.w) + (BLUE_COEFFICIENT * blue.w);

	// Store 4 pixels back to the device buffer
	((uchar4*)grayImage)[(y * quarterPitch) + x] = grey;
}

void rgb2gray(uchar *inputImage, uchar *grayImage, const int width, const int height, NSTimer &timer) {
	hipError_t error = hipSuccess;

	// Force the initialization of the device context to make sure the timers are accurate
	error = hipFree(0);
	checkError(error, "Unable to initialize device context (error code %s)\n");

	// Initialize timers
	NSTimer allocationTime = NSTimer("allocateTime", false, false);
	NSTimer copyToDeviceTime = NSTimer("copyToDeviceTime", false, false);
	NSTimer kernelTime = NSTimer("kernelTime", false, false);
	NSTimer copyFromDeviceTime = NSTimer("copyFromDeviceTime", false, false);

	// Allocate two device buffers
	allocationTime.start();
	uchar *inputImage_device, *grayImage_device;
	size_t pitch;
	error = hipMallocPitch(&inputImage_device, &pitch, width * sizeof(uchar), height * 3);
	checkError(error, "Failed to allocate device buffer inputImage_device (error code %s)\n");
	error = hipMallocPitch(&grayImage_device, &pitch, width * sizeof(uchar), height);
	checkError(error, "Failed to allocate device buffer grayImage_device (error code %s)\n");
	allocationTime.stop();

	// Copy the input image from the host to the device
	copyToDeviceTime.start();
	error = hipMemcpy2D(inputImage_device, pitch, inputImage, width * sizeof(uchar), width * sizeof(uchar), height * 3, hipMemcpyHostToDevice);
	checkError(error, "Failed to copy inputImage from host to device (error code %s)\n");
	copyToDeviceTime.stop();

	// Launch the kernel
	kernelTime.start();
	dim3 threadsPerBlock(16, 16);
	dim3 blocksPerGrid(ceil((float)width / 4 / threadsPerBlock.x), ceil((float)height / threadsPerBlock.y));
	rgb2gray_kernel<<<blocksPerGrid, threadsPerBlock>>>(inputImage_device, grayImage_device, width, height, pitch);
	checkError(hipGetLastError(), "Failed to launch rgb2gray_kernel (error code %s)\n");
	hipDeviceSynchronize();
	kernelTime.stop();

	// Copy the grayscale image from the device to the host
	copyFromDeviceTime.start();
	error = hipMemcpy2D(grayImage, width, grayImage_device, pitch, width * sizeof(uchar), height, hipMemcpyDeviceToHost);
	checkError(error, "Failed to copy grayImage from device to host (error code %s)\n");
	copyFromDeviceTime.stop();

	// Free the device buffers
	hipFree(inputImage_device);
	hipFree(grayImage_device);

	// Print the timers
	cout << fixed << setprecision(6);
	//cout << "rgb2gray (allocation): \t\t\t" << allocationTime.getElapsed() << " seconds." << endl;
	//cout << "rgb2gray (copyToDevice): \t\t" << copyToDeviceTime.getElapsed() << " seconds." << endl;
	cout << "rgb2gray (kernel): \t\t\t" << kernelTime.getElapsed() << " seconds." << endl;
	//cout << "rgb2gray (copyFromDevice): \t\t" << copyFromDeviceTime.getElapsed() << " seconds." << endl;
}

__global__ void
histogram1D_kernel(uchar *grayImage, const int width, const int height, uint *histogram, const size_t pitch) {
	const int x = blockDim.x * blockIdx.x + threadIdx.x;
	const int y = blockDim.y * blockIdx.y + threadIdx.y;

	// Fetch 4 bytes from device buffer
	const uchar4 in = ((uchar4*)grayImage)[(y * pitch / 4) + x];

	// Initialize shared histogram
	const int histogram_index = blockDim.y * threadIdx.y + threadIdx.x;
	__shared__ uchar histogram_shared[HISTOGRAM_SIZE];
	histogram_shared[histogram_index] = 0;
	__syncthreads();

	// Make sure we are within bounds
	if (x >= width || y >= height) return;

	// Add pixel data to shared histogram
	histogram_shared[in.x]++;
	histogram_shared[in.y]++;
	histogram_shared[in.z]++;
	histogram_shared[in.w]++;

	// Atomically add shared histogram to global histogram
	__syncthreads();
	atomicAdd(&histogram[histogram_index], histogram_shared[histogram_index]);
}

void histogram1D(uchar *grayImage, uchar *histogramImage, const int width, const int height, uint *histogram, NSTimer &timer) {
	hipError_t error = hipSuccess;

	// Initialize timers
	NSTimer allocationTime = NSTimer("allocateTime", false, false);
	NSTimer copyToDeviceTime = NSTimer("copyToDeviceTime", false, false);
	NSTimer kernelTime = NSTimer("kernelTime", false, false);
	NSTimer copyFromDeviceTime = NSTimer("copyFromDeviceTime", false, false);

	// Allocate two device buffers
	allocationTime.start();
	uchar *grayImage_device;
	uint *histogram_device;
	size_t pitch;
	error = hipMallocPitch(&grayImage_device, &pitch, width * sizeof(uchar), height);
	checkError(error, "Failed to allocate device buffer grayImage_device (error code %s)\n");
	error = hipMalloc(&histogram_device, HISTOGRAM_SIZE * sizeof(uint));
	checkError(error, "Failed to allocate device buffer histogram_device (error code %s)\n");
	allocationTime.stop();

	// Set histogram buffer to 0
	error = hipMemset(reinterpret_cast< void * >(histogram_device), 0, HISTOGRAM_SIZE * sizeof(uint));
	checkError(error, "Failed to set histogram buffer to 0 (error code %s)\n");

	// Copy the grayscale image from the host to the device
	copyToDeviceTime.start();
	error = hipMemcpy2D(grayImage_device, pitch, grayImage, width * sizeof(uchar), width * sizeof(uchar), height, hipMemcpyHostToDevice);
	checkError(error, "Failed to copy grayImage from host to device (error code %s)\n");
	copyToDeviceTime.stop();

	// Launch the kernel
	kernelTime.start();
	dim3 threadsPerBlock(16, 16); // Product must be 256
	dim3 blocksPerGrid(ceil((float)width / 4 / threadsPerBlock.x), ceil((float)height / threadsPerBlock.y));
	histogram1D_kernel<<<blocksPerGrid, threadsPerBlock>>>(grayImage_device, width, height, histogram_device, pitch);
	checkError(hipGetLastError(), "Failed to launch histogram1D_kernel (error code %s)\n");
	hipDeviceSynchronize();
	kernelTime.stop();

	// Copy the histogram from the device to the host
	copyFromDeviceTime.start();
	error = hipMemcpy(histogram, histogram_device, HISTOGRAM_SIZE * sizeof(uint), hipMemcpyDeviceToHost);
	checkError(error, "Failed to copy histogram from device to host (error code %s)\n");
	copyFromDeviceTime.stop();

	// Free the device buffers
	hipFree(grayImage_device);
	hipFree(histogram_device);

	// Find maximum in histogram
	uint max = 0;
	for ( uint i = 0; i < HISTOGRAM_SIZE; i++ ) {
		if ( histogram[i] > max ) {
			max = histogram[i];
		}
	}

	// Generate histogram image
	for ( int x = 0; x < HISTOGRAM_SIZE * BAR_WIDTH; x += BAR_WIDTH ) {
		uint value = HISTOGRAM_SIZE - ((histogram[x / BAR_WIDTH] * HISTOGRAM_SIZE) / max);

		for ( uint y = 0; y < value; y++ ) {
			for ( uint i = 0; i < BAR_WIDTH; i++ ) {
				histogramImage[(y * HISTOGRAM_SIZE * BAR_WIDTH) + x + i] = 0;
			}
		}
		for ( uint y = value; y < HISTOGRAM_SIZE; y++ ) {
			for ( uint i = 0; i < BAR_WIDTH; i++ ) {
				histogramImage[(y * HISTOGRAM_SIZE * BAR_WIDTH) + x + i] = 255;
			}
		}
	}

	// Print the timers
	cout << fixed << setprecision(6);
	//cout << "histogram1D (allocation): \t\t" << allocationTime.getElapsed() << " seconds." << endl;
	//cout << "histogram1D (copyToDevice): \t\t" << copyToDeviceTime.getElapsed() << " seconds." << endl;
	cout << "histogram1D (kernel): \t\t\t" << kernelTime.getElapsed() << " seconds." << endl;
	//cout << "histogram1D (copyFromDevice): \t\t" << copyFromDeviceTime.getElapsed() << " seconds." << endl;
}

__device__ void
contrast1D_pixelValue (uchar &pixel, const uint min, const uint diff) {
	float temp = 255.0f * (pixel - min) / diff;
	temp = fminf(temp, 255.0f);
	temp = fmaxf(temp, 0.0f);
	pixel = static_cast< uchar >(temp);
}

__global__ void
contrast1D_kernel(uchar *grayImage, const int width, const int height, const uint min, const uint diff, const size_t pitch) {
	const int x = blockDim.x * blockIdx.x + threadIdx.x;
	const int y = blockDim.y * blockIdx.y + threadIdx.y;

	// Fetch 4 pixels from device buffer
	uchar4 pixels = ((uchar4*)grayImage)[(y * pitch / 4) + x];

	// Make sure we are within bounds
	if (x >= width || y >= height) return;

	contrast1D_pixelValue(pixels.x, min, diff);
	contrast1D_pixelValue(pixels.y, min, diff);
	contrast1D_pixelValue(pixels.z, min, diff);
	contrast1D_pixelValue(pixels.w, min, diff);

	// Store 4 pixels back to the device buffer
	((uchar4*)grayImage)[(y * pitch / 4) + x] = pixels;
}

void contrast1D(uchar *grayImage, const int width, const int height, uint *histogram, NSTimer &timer) {
	hipError_t error = hipSuccess;

	// Initialize timers
	NSTimer allocationTime = NSTimer("allocateTime", false, false);
	NSTimer copyToDeviceTime = NSTimer("copyToDeviceTime", false, false);
	NSTimer kernelTime = NSTimer("kernelTime", false, false);
	NSTimer copyFromDeviceTime = NSTimer("copyFromDeviceTime", false, false);

	// Allocate device buffer
	allocationTime.start();
	uchar *grayImage_device;
	size_t pitch;
	error = hipMallocPitch(&grayImage_device, &pitch, width * sizeof(uchar), height);
	checkError(error, "Failed to allocate device buffer grayImage_device (error code %s)\n");
	allocationTime.stop();

	// Determine minimum, maximum and their difference of histogram
	uint i = 0;

	while ( (i < HISTOGRAM_SIZE) && (histogram[i] < CONTRAST_THRESHOLD) ) {
		i++;
	}
	uint min = i;

	i = HISTOGRAM_SIZE - 1;
	while ( (i > min) && (histogram[i] < CONTRAST_THRESHOLD) ) {
		i--;
	}
	uint max = i;
	uint diff = max - min;

	// Copy the grayscale image from the host to the device
	copyToDeviceTime.start();
	error = hipMemcpy2D(grayImage_device, pitch, grayImage, width * sizeof(uchar), width * sizeof(uchar), height, hipMemcpyHostToDevice);
	checkError(error, "Failed to copy grayImage from host to device (error code %s)\n");
	copyToDeviceTime.stop();

	// Launch the kernel
	kernelTime.start();
	dim3 threadsPerBlock(16, 16);
	dim3 blocksPerGrid(ceil((float)width / 4 / threadsPerBlock.x), ceil((float)height / threadsPerBlock.y));
	contrast1D_kernel<<<blocksPerGrid, threadsPerBlock>>>(grayImage_device, width, height, min, diff, pitch);
	checkError(hipGetLastError(), "Failed to launch contrast1D_kernel (error code %s)\n");
	hipDeviceSynchronize();
	kernelTime.stop();

	// Copy the grayscale image from the device to the host
	copyFromDeviceTime.start();
	error = hipMemcpy2D(grayImage, width, grayImage_device, pitch, width * sizeof(uchar), height, hipMemcpyDeviceToHost);
	checkError(error, "Failed to copy grayImage from device to host (error code %s)\n");
	copyFromDeviceTime.stop();

	// Free the device buffer
	hipFree(grayImage_device);

	// Print the timers
	cout << fixed << setprecision(6);
	//cout << "contrast1D (allocation): \t\t" << allocationTime.getElapsed() << " seconds." << endl;
	//cout << "contrast1D (copyToDevice): \t\t" << copyToDeviceTime.getElapsed() << " seconds." << endl;
	cout << "contrast1D (kernel): \t\t\t" << kernelTime.getElapsed() << " seconds." << endl;
	//cout << "contrast1D (copyFromDevice): \t\t" << copyFromDeviceTime.getElapsed() << " seconds." << endl;
}

__constant__ float filter_constant[FILTER_SIZE][FILTER_SIZE];

__global__ void
triangularSmooth_kernel(uchar *grayImage, uchar *smoothImage, const int width, const int height) {
	const int x = blockDim.x * blockIdx.x + threadIdx.x + 2;
	const int y = blockDim.y * blockIdx.y + threadIdx.y + 2;

	__shared__ uchar grayImage_shared[(SMOOTH_BLOCK_WIDTH + 4) * (SMOOTH_BLOCK_HEIGHT + 4)];

	grayImage_shared[(SMOOTH_BLOCK_WIDTH + 4) * (threadIdx.y + 2) + (threadIdx.x + 2)] = grayImage[y * width + x];

	if (threadIdx.x == 0) {
		grayImage_shared[(SMOOTH_BLOCK_WIDTH + 4) * (threadIdx.y + 2) + (threadIdx.x + 0)] = grayImage[(y + 0) * width + (x - 2)];
		grayImage_shared[(SMOOTH_BLOCK_WIDTH + 4) * (threadIdx.y + 2) + (threadIdx.x + 1)] = grayImage[(y + 0) * width + (x - 1)];
	}

	if (threadIdx.y == 0) {
		grayImage_shared[(SMOOTH_BLOCK_WIDTH + 4) * (threadIdx.y + 0) + (threadIdx.x + 2)] = grayImage[(y - 2) * width + (x + 0)];
		grayImage_shared[(SMOOTH_BLOCK_WIDTH + 4) * (threadIdx.y + 1) + (threadIdx.x + 2)] = grayImage[(y - 1) * width + (x + 0)];
	}

	if (threadIdx.x == SMOOTH_BLOCK_WIDTH - 1) {
		grayImage_shared[(SMOOTH_BLOCK_WIDTH + 4) * (threadIdx.y + 2) + (threadIdx.x + 3)] = x + 1 < width ? grayImage[(y + 0) * width + (x + 1)] : 0;
		grayImage_shared[(SMOOTH_BLOCK_WIDTH + 4) * (threadIdx.y + 2) + (threadIdx.x + 4)] = x + 2 < width ? grayImage[(y + 0) * width + (x + 2)] : 0;
	}

	if (threadIdx.y == SMOOTH_BLOCK_HEIGHT - 1) {
		grayImage_shared[(SMOOTH_BLOCK_WIDTH + 4) * (threadIdx.y + 3) + (threadIdx.x + 2)] = y + 1 < height ? grayImage[(y + 1) * width + (x + 0)] : 0;
		grayImage_shared[(SMOOTH_BLOCK_WIDTH + 4) * (threadIdx.y + 4) + (threadIdx.x + 2)] = y + 2 < height ? grayImage[(y + 2) * width + (x + 0)] : 0;
	}

	if (threadIdx.x == 0 && threadIdx.y == 0) {
		grayImage_shared[(SMOOTH_BLOCK_WIDTH + 4) * (threadIdx.y + 0) + (threadIdx.x + 0)] = grayImage[(y - 2) * width + (x - 2)];
		grayImage_shared[(SMOOTH_BLOCK_WIDTH + 4) * (threadIdx.y + 0) + (threadIdx.x + 1)] = grayImage[(y - 2) * width + (x - 1)];
		grayImage_shared[(SMOOTH_BLOCK_WIDTH + 4) * (threadIdx.y + 1) + (threadIdx.x + 0)] = grayImage[(y - 1) * width + (x - 2)];
		grayImage_shared[(SMOOTH_BLOCK_WIDTH + 4) * (threadIdx.y + 1) + (threadIdx.x + 1)] = grayImage[(y - 1) * width + (x - 1)];
	}

	if (threadIdx.x == SMOOTH_BLOCK_WIDTH - 1 && threadIdx.y == 0) {
		grayImage_shared[(SMOOTH_BLOCK_WIDTH + 4) * (threadIdx.y + 0) + (threadIdx.x + 3)] = x + 1 < width ? grayImage[(y - 2) * width + (x + 1)] : 0;
		grayImage_shared[(SMOOTH_BLOCK_WIDTH + 4) * (threadIdx.y + 0) + (threadIdx.x + 4)] = x + 2 < width ? grayImage[(y - 2) * width + (x + 2)] : 0;
		grayImage_shared[(SMOOTH_BLOCK_WIDTH + 4) * (threadIdx.y + 1) + (threadIdx.x + 3)] = x + 1 < width ? grayImage[(y - 1) * width + (x + 1)] : 0;
		grayImage_shared[(SMOOTH_BLOCK_WIDTH + 4) * (threadIdx.y + 1) + (threadIdx.x + 4)] = x + 2 < width ? grayImage[(y - 1) * width + (x + 2)] : 0;
	}

	if (threadIdx.x == 0 && threadIdx.y == SMOOTH_BLOCK_HEIGHT - 1) {
		grayImage_shared[(SMOOTH_BLOCK_WIDTH + 4) * (threadIdx.y + 3) + (threadIdx.x + 0)] = y + 1 < height ? grayImage[(y + 1) * width + (x - 2)] : 0;
		grayImage_shared[(SMOOTH_BLOCK_WIDTH + 4) * (threadIdx.y + 3) + (threadIdx.x + 1)] = y + 1 < height ? grayImage[(y + 1) * width + (x - 1)] : 0;
		grayImage_shared[(SMOOTH_BLOCK_WIDTH + 4) * (threadIdx.y + 4) + (threadIdx.x + 0)] = y + 2 < height ? grayImage[(y + 2) * width + (x - 2)] : 0;
		grayImage_shared[(SMOOTH_BLOCK_WIDTH + 4) * (threadIdx.y + 4) + (threadIdx.x + 1)] = y + 2 < height ? grayImage[(y + 2) * width + (x - 1)] : 0;
	}

	if (threadIdx.x == SMOOTH_BLOCK_WIDTH - 1 && threadIdx.y == SMOOTH_BLOCK_HEIGHT - 1) {
		grayImage_shared[(SMOOTH_BLOCK_WIDTH + 4) * (threadIdx.y + 3) + (threadIdx.x + 3)] = y + 1 < height && x + 1 < width ? grayImage[(y + 1) * width + (x + 1)] : 0;
		grayImage_shared[(SMOOTH_BLOCK_WIDTH + 4) * (threadIdx.y + 3) + (threadIdx.x + 4)] = y + 1 < height && x + 2 < width ? grayImage[(y + 1) * width + (x + 2)] : 0;
		grayImage_shared[(SMOOTH_BLOCK_WIDTH + 4) * (threadIdx.y + 4) + (threadIdx.x + 3)] = y + 2 < height && x + 1 < width ? grayImage[(y + 2) * width + (x + 1)] : 0;
		grayImage_shared[(SMOOTH_BLOCK_WIDTH + 4) * (threadIdx.y + 4) + (threadIdx.x + 4)] = y + 2 < height && x + 2 < width ? grayImage[(y + 2) * width + (x + 2)] : 0;
	}
	
	__syncthreads();

	// Make sure we are within bounds
	if (x >= width - 2 || y >= height - 2) return;

	float filterSum = 0.0f;
	float smoothPix = 0.0f;

	for ( int dy = -2; dy <= 2; dy++ ) {
		for ( int dx = -2; dx <= 2; dx++ ) {
			smoothPix += grayImage_shared[(threadIdx.y + dy + 2) * (SMOOTH_BLOCK_WIDTH + 4) + (threadIdx.x + dx + 2)] * filter_constant[dy+2][dx+2];
			filterSum += filter_constant[dy+2][dx+2];
		}
	}

	smoothPix /= filterSum;
	smoothImage[(y * width) + x] = static_cast< uchar >(smoothPix);
}

void triangularSmooth(uchar *grayImage, uchar *smoothImage, const int width, const int height, const float *filter, NSTimer &timer) {
	hipError_t error = hipSuccess;

	// Initialize timers
	NSTimer allocationTime = NSTimer("allocateTime", false, false);
	NSTimer copyToDeviceTime = NSTimer("copyToDeviceTime", false, false);
	NSTimer kernelTime = NSTimer("kernelTime", false, false);
	NSTimer copyFromDeviceTime = NSTimer("copyFromDeviceTime", false, false);

	// Allocate three device buffers
	allocationTime.start();
	uchar *grayImage_device, *smoothImage_device;
	error = hipMalloc(&grayImage_device, width * height * sizeof(uchar));
	checkError(error, "Failed to allocate device buffer grayImage_device (error code %s)\n");
	error = hipMalloc(&smoothImage_device, width * height * sizeof(uchar));
	checkError(error, "Failed to allocate device buffer smoothImage_device (error code %s)\n");
	allocationTime.stop();

	// Copy the grayscale image and the filter from the host to the device
	copyToDeviceTime.start();
	error = hipMemcpy(grayImage_device, grayImage, width * height * sizeof(uchar), hipMemcpyHostToDevice);
	checkError(error, "Failed to copy grayImage from host to device (error code %s)\n");
	error = hipMemcpyToSymbol(HIP_SYMBOL(filter_constant), filter, FILTER_SIZE * FILTER_SIZE * sizeof(float));
	checkError(error, "Failed to copy filter from host to device (error code %s)\n");
	copyToDeviceTime.stop();

	// Launch the kernel
	kernelTime.start();
	dim3 threadsPerBlock(SMOOTH_BLOCK_WIDTH, SMOOTH_BLOCK_HEIGHT);
	dim3 blocksPerGrid(ceil((float)(width - 4) / threadsPerBlock.x), ceil((float)(height - 4) / threadsPerBlock.y));
	triangularSmooth_kernel<<<blocksPerGrid, threadsPerBlock>>>(grayImage_device, smoothImage_device, width, height);
	checkError(hipGetLastError(), "Failed to launch triangularSmooth_kernel (error code %s)\n");
	hipDeviceSynchronize();
	kernelTime.stop();

	// Copy the smooth image from the device to the host
	copyFromDeviceTime.start();
	error = hipMemcpy(smoothImage, smoothImage_device, width * height * sizeof(uchar), hipMemcpyDeviceToHost);
	checkError(error, "Failed to copy smoothImage from device to host (error code %s)\n");
	copyFromDeviceTime.stop();

	// Free the device buffers
	hipFree(grayImage_device);
	hipFree(smoothImage_device);

	// Print the timers
	cout << fixed << setprecision(6);
	//cout << "triangularSmooth (allocation): \t\t" << allocationTime.getElapsed() << " seconds." << endl;
	//cout << "triangularSmooth (copyToDevice): \t" << copyToDeviceTime.getElapsed() << " seconds." << endl;
	cout << "triangularSmooth (kernel): \t\t" << kernelTime.getElapsed() << " seconds." << endl;
	//cout << "triangularSmooth (copyFromDevice): \t" << copyFromDeviceTime.getElapsed() << " seconds." << endl;
}

