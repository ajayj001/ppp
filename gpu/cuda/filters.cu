#include "hip/hip_runtime.h"

#include <Timer.hpp>
#include <iostream>
#include <iomanip>
#include "defines.h"

using LOFAR::NSTimer;
using std::cout;
using std::cerr;
using std::endl;
using std::fixed;
using std::setprecision;


void checkError(hipError_t error, const char* description) {
	if (error != hipSuccess) {
		fprintf(stderr, description, hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
}

__global__ void
rgb2gray_kernel(uchar *inputImage, uchar *grayImage, const int width, const int height, const size_t pitch) {
	const int x = blockDim.x * blockIdx.x + threadIdx.x;
	const int y = blockDim.y * blockIdx.y + threadIdx.y;

	// Make sure we are within bounds
	if (x >= width || y >= height) return;

	float r = static_cast< float >(inputImage[(y * pitch) + x]);
	float g = static_cast< float >(inputImage[(pitch * height) + (y * pitch) + x]);
	float b = static_cast< float >(inputImage[(2 * pitch * height) + (y * pitch) + x]);

	float grayPix = (0.3f * r) + (0.59f * g) + (0.11f * b);

	grayImage[(y * pitch) + x] = static_cast< uchar >(grayPix);
}

void rgb2gray(uchar *inputImage, uchar *grayImage, const int width, const int height, NSTimer &timer) {
	hipError_t error = hipSuccess;

	// Force the initialization of the device context to make sure the timers are accurate
	error = hipFree(0);
	checkError(error, "Unable to initialize device context (error code %s)\n");

	// Initialize timers
	NSTimer allocationTime = NSTimer("allocateTime", false, false);
	NSTimer copyToDeviceTime = NSTimer("copyToDeviceTime", false, false);
	NSTimer kernelTime = NSTimer("kernelTime", false, false);
	NSTimer copyFromDeviceTime = NSTimer("copyFromDeviceTime", false, false);

	// Allocate two device buffers
	allocationTime.start();
	uchar *inputImage_device, *grayImage_device;
	size_t pitch;
	error = hipMallocPitch(&inputImage_device, &pitch, width * sizeof(uchar), height * 3);
	checkError(error, "Failed to allocate device buffer inputImage_device (error code %s)\n");
	error = hipMallocPitch(&grayImage_device, &pitch, width * sizeof(uchar), height);
	checkError(error, "Failed to allocate device buffer grayImage_device (error code %s)\n");
	allocationTime.stop();

	// Copy the input image from the host to the device
	copyToDeviceTime.start();
	error = hipMemcpy2D(inputImage_device, pitch, inputImage, width * sizeof(uchar), width * sizeof(uchar), height * 3, hipMemcpyHostToDevice);
	checkError(error, "Failed to copy inputImage from host to device (error code %s)\n");
	copyToDeviceTime.stop();

	// Launch the kernel
	kernelTime.start();
	dim3 threadsPerBlock(128, 4);
	dim3 blocksPerGrid(ceil((float)width / threadsPerBlock.x), ceil((float)height / threadsPerBlock.y));
	rgb2gray_kernel<<<blocksPerGrid, threadsPerBlock>>>(inputImage_device, grayImage_device, width, height, pitch);
	checkError(hipGetLastError(), "Failed to launch rgb2gray_kernel (error code %s)\n");
	hipDeviceSynchronize();
	kernelTime.stop();

	// Copy the grayscale image from the device to the host
	copyFromDeviceTime.start();
	error = hipMemcpy2D(grayImage, width, grayImage_device, pitch, width * sizeof(uchar), height, hipMemcpyDeviceToHost);
	checkError(error, "Failed to copy grayImage from device to host (error code %s)\n");
	copyFromDeviceTime.stop();

	// Free the device buffers
	hipFree(inputImage_device);
	hipFree(grayImage_device);

	// Print the timers
	cout << fixed << setprecision(6);
	//cout << "rgb2gray (allocation): \t\t\t" << allocationTime.getElapsed() << " seconds." << endl;
	//cout << "rgb2gray (copyToDevice): \t\t" << copyToDeviceTime.getElapsed() << " seconds." << endl;
	cout << "rgb2gray (kernel): \t\t\t" << kernelTime.getElapsed() << " seconds." << endl;
	//cout << "rgb2gray (copyFromDevice): \t\t" << copyFromDeviceTime.getElapsed() << " seconds." << endl;
}

__global__ void
histogram1D_kernel(uchar *grayImage, const int width, const int height, uint *histogram, const size_t pitch) {
	const int x_base = (blockDim.x * blockIdx.x + threadIdx.x) * HISTOGRAM_PIXELS_WIDTH;
	const int y_base = (blockDim.y * blockIdx.y + threadIdx.y) * HISTOGRAM_PIXELS_HEIGHT;
	const int histogram_index = blockDim.y * threadIdx.y + threadIdx.x;

	// Initialize shared histogram
	__shared__ uchar histogram_shared[HISTOGRAM_SIZE];
	histogram_shared[histogram_index] = 0;

	for (int x = x_base; x < x_base + HISTOGRAM_PIXELS_WIDTH; x++) {
		for (int y = y_base; y < y_base + HISTOGRAM_PIXELS_HEIGHT; y++) {
			// Make sure we are within bounds
			if (x >= width || y >= height) continue;

			// Add pixel data to shared histogram
			histogram_shared[static_cast< uint >(grayImage[(y * pitch) + x])]++;
		}
	}

	// Atomically add shared histogram to global histogram
	__syncthreads();
	atomicAdd(&histogram[histogram_index], histogram_shared[histogram_index]);
}

void histogram1D(uchar *grayImage, uchar *histogramImage, const int width, const int height, uint *histogram, NSTimer &timer) {
	hipError_t error = hipSuccess;

	// Initialize timers
	NSTimer allocationTime = NSTimer("allocateTime", false, false);
	NSTimer copyToDeviceTime = NSTimer("copyToDeviceTime", false, false);
	NSTimer kernelTime = NSTimer("kernelTime", false, false);
	NSTimer copyFromDeviceTime = NSTimer("copyFromDeviceTime", false, false);

	// Allocate two device buffers
	allocationTime.start();
	uchar *grayImage_device;
	uint *histogram_device;
	size_t pitch;
	error = hipMallocPitch(&grayImage_device, &pitch, width * sizeof(uchar), height);
	checkError(error, "Failed to allocate device buffer grayImage_device (error code %s)\n");
	error = hipMalloc(&histogram_device, HISTOGRAM_SIZE * sizeof(uint));
	checkError(error, "Failed to allocate device buffer histogram_device (error code %s)\n");
	allocationTime.stop();

	// Set histogram buffer to 0
	error = hipMemset(reinterpret_cast< void * >(histogram_device), 0, HISTOGRAM_SIZE * sizeof(uint));
	checkError(error, "Failed to set histogram buffer to 0 (error code %s)\n");

	// Copy the grayscale image from the host to the device
	copyToDeviceTime.start();
	error = hipMemcpy2D(grayImage_device, pitch, grayImage, width * sizeof(uchar), width * sizeof(uchar), height, hipMemcpyHostToDevice);
	checkError(error, "Failed to copy grayImage from host to device (error code %s)\n");
	copyToDeviceTime.stop();

	// Launch the kernel
	kernelTime.start();
	dim3 threadsPerBlock(16, 16); // Product must be 256
	dim3 blocksPerGrid(ceil((float)width / HISTOGRAM_PIXELS_WIDTH / threadsPerBlock.x), ceil((float)height / HISTOGRAM_PIXELS_HEIGHT / threadsPerBlock.y));
	histogram1D_kernel<<<blocksPerGrid, threadsPerBlock>>>(grayImage_device, width, height, histogram_device, pitch);
	checkError(hipGetLastError(), "Failed to launch histogram1D_kernel (error code %s)\n");
	hipDeviceSynchronize();
	kernelTime.stop();

	// Copy the histogram from the device to the host
	copyFromDeviceTime.start();
	error = hipMemcpy(histogram, histogram_device, HISTOGRAM_SIZE * sizeof(uint), hipMemcpyDeviceToHost);
	checkError(error, "Failed to copy histogram from device to host (error code %s)\n");
	copyFromDeviceTime.stop();

	// Free the device buffers
	hipFree(grayImage_device);
	hipFree(histogram_device);

	// Find maximum in histogram
	uint max = 0;
	for ( uint i = 0; i < HISTOGRAM_SIZE; i++ ) {
		if ( histogram[i] > max ) {
			max = histogram[i];
		}
	}

	// Generate histogram image
	for ( int x = 0; x < HISTOGRAM_SIZE * BAR_WIDTH; x += BAR_WIDTH ) {
		uint value = HISTOGRAM_SIZE - ((histogram[x / BAR_WIDTH] * HISTOGRAM_SIZE) / max);

		for ( uint y = 0; y < value; y++ ) {
			for ( uint i = 0; i < BAR_WIDTH; i++ ) {
				histogramImage[(y * HISTOGRAM_SIZE * BAR_WIDTH) + x + i] = 0;
			}
		}
		for ( uint y = value; y < HISTOGRAM_SIZE; y++ ) {
			for ( uint i = 0; i < BAR_WIDTH; i++ ) {
				histogramImage[(y * HISTOGRAM_SIZE * BAR_WIDTH) + x + i] = 255;
			}
		}
	}

	// Print the timers
	cout << fixed << setprecision(6);
	//cout << "histogram1D (allocation): \t\t" << allocationTime.getElapsed() << " seconds." << endl;
	//cout << "histogram1D (copyToDevice): \t\t" << copyToDeviceTime.getElapsed() << " seconds." << endl;
	cout << "histogram1D (kernel): \t\t\t" << kernelTime.getElapsed() << " seconds." << endl;
	//cout << "histogram1D (copyFromDevice): \t\t" << copyFromDeviceTime.getElapsed() << " seconds." << endl;
}

__global__ void
contrast1D_kernel(uchar *grayImage, const int width, const int height, const uint min, const uint max, const float diff, const size_t pitch) {
	const int x_base = blockDim.x * blockIdx.x + threadIdx.x;
	const int y = blockDim.y * blockIdx.y + threadIdx.y;

	for (int x = x_base; x < x_base + CONTRAST1D_PIXELS_PER_THREAD; x++) {
		// Make sure we are within bounds
		if (x >= width || y >= height) continue;

		uchar pixel = grayImage[(y * pitch) + x];

		if ( pixel < min ) {
			pixel = 0;
		}
		else if ( pixel > max ) {
			pixel = 255;
		}
		else {
			pixel = static_cast< uchar >(255.0f * (pixel - min) / diff);
		}

		grayImage[(y * pitch) + x] = pixel;
	}
}

void contrast1D(uchar *grayImage, const int width, const int height, uint *histogram, NSTimer &timer) {
	hipError_t error = hipSuccess;

	// Initialize timers
	NSTimer allocationTime = NSTimer("allocateTime", false, false);
	NSTimer copyToDeviceTime = NSTimer("copyToDeviceTime", false, false);
	NSTimer kernelTime = NSTimer("kernelTime", false, false);
	NSTimer copyFromDeviceTime = NSTimer("copyFromDeviceTime", false, false);

	// Allocate device buffer
	allocationTime.start();
	uchar *grayImage_device;
	size_t pitch;
	error = hipMallocPitch(&grayImage_device, &pitch, width * sizeof(uchar), height);
	checkError(error, "Failed to allocate device buffer grayImage_device (error code %s)\n");
	allocationTime.stop();

	// Determine minimum, maximum and their difference of histogram
	uint i = 0;

	while ( (i < HISTOGRAM_SIZE) && (histogram[i] < CONTRAST_THRESHOLD) ) {
		i++;
	}
	uint min = i;

	i = HISTOGRAM_SIZE - 1;
	while ( (i > min) && (histogram[i] < CONTRAST_THRESHOLD) ) {
		i--;
	}
	uint max = i;
	float diff = max - min;

	// Copy the grayscale image from the host to the device
	copyToDeviceTime.start();
	error = hipMemcpy2D(grayImage_device, pitch, grayImage, width * sizeof(uchar), width * sizeof(uchar), height, hipMemcpyHostToDevice);
	checkError(error, "Failed to copy grayImage from host to device (error code %s)\n");
	copyToDeviceTime.stop();

	// Launch the kernel
	kernelTime.start();
	dim3 threadsPerBlock(16, 16);
	dim3 blocksPerGrid(ceil((float)width / CONTRAST1D_PIXELS_PER_THREAD / threadsPerBlock.x), ceil((float)height / threadsPerBlock.y));
	contrast1D_kernel<<<blocksPerGrid, threadsPerBlock>>>(grayImage_device, width, height, min, max, diff, pitch);
	checkError(hipGetLastError(), "Failed to launch contrast1D_kernel (error code %s)\n");
	hipDeviceSynchronize();
	kernelTime.stop();

	// Copy the grayscale image from the device to the host
	copyFromDeviceTime.start();
	error = hipMemcpy2D(grayImage, width, grayImage_device, pitch, width * sizeof(uchar), height, hipMemcpyDeviceToHost);
	checkError(error, "Failed to copy grayImage from device to host (error code %s)\n");
	copyFromDeviceTime.stop();

	// Free the device buffer
	hipFree(grayImage_device);

	// Print the timers
	cout << fixed << setprecision(6);
	//cout << "contrast1D (allocation): \t\t" << allocationTime.getElapsed() << " seconds." << endl;
	//cout << "contrast1D (copyToDevice): \t\t" << copyToDeviceTime.getElapsed() << " seconds." << endl;
	cout << "contrast1D (kernel): \t\t\t" << kernelTime.getElapsed() << " seconds." << endl;
	//cout << "contrast1D (copyFromDevice): \t\t" << copyFromDeviceTime.getElapsed() << " seconds." << endl;
}

__constant__ float filter_constant[FILTER_LENGTH];

__global__ void
triangularSmooth_kernel(uchar *grayImage, uchar *smoothImage, const int width, const int height, const size_t pitch) {
	const int x = blockDim.x * blockIdx.x + threadIdx.x;
	const int y = blockDim.y * blockIdx.y + threadIdx.y;

	// Make sure we are within bounds
	if (x >= width || y >= height) return;

	uint filterItem = 0;
	float filterSum = 0.0f;
	float smoothPix = 0.0f;

	for ( int fy = y - 2; fy < y + 3; fy++ ) {
		for ( int fx = x - 2; fx < x + 3; fx++ ) {
			if ( ((fy < 0) || (fy >= height)) || ((fx < 0) || (fx >= width)) ) {
				filterItem++;
				continue;
			}

			smoothPix += grayImage[(fy * pitch) + fx] * filter_constant[filterItem];
			filterSum += filter_constant[filterItem];
			filterItem++;
		}
	}

	smoothPix /= filterSum;
	smoothImage[(y * pitch) + x] = static_cast< uchar >(smoothPix);
}

void triangularSmooth(uchar *grayImage, uchar *smoothImage, const int width, const int height, const float *filter, NSTimer &timer) {
	hipError_t error = hipSuccess;

	// Initialize timers
	NSTimer allocationTime = NSTimer("allocateTime", false, false);
	NSTimer copyToDeviceTime = NSTimer("copyToDeviceTime", false, false);
	NSTimer kernelTime = NSTimer("kernelTime", false, false);
	NSTimer copyFromDeviceTime = NSTimer("copyFromDeviceTime", false, false);

	// Allocate three device buffers
	allocationTime.start();
	uchar *grayImage_device, *smoothImage_device;
	size_t pitch;
	error = hipMallocPitch(&grayImage_device, &pitch, width * sizeof(uchar), height);
	checkError(error, "Failed to allocate device buffer grayImage_device (error code %s)\n");
	error = hipMallocPitch(&smoothImage_device, &pitch, width * sizeof(uchar), height);
	checkError(error, "Failed to allocate device buffer smoothImage_device (error code %s)\n");
	allocationTime.stop();

	// Copy the grayscale image and the filter from the host to the device
	copyToDeviceTime.start();
	error = hipMemcpy2D(grayImage_device, pitch, grayImage, width * sizeof(uchar), width * sizeof(uchar), height, hipMemcpyHostToDevice);
	checkError(error, "Failed to copy grayImage from host to device (error code %s)\n");
	error = hipMemcpyToSymbol(HIP_SYMBOL(filter_constant), filter, FILTER_LENGTH * sizeof(float));
	checkError(error, "Failed to copy filter from host to device (error code %s)\n");
	copyToDeviceTime.stop();

	// Launch the kernel
	kernelTime.start();
	dim3 threadsPerBlock(16, 16);
	dim3 blocksPerGrid(ceil((float)width / threadsPerBlock.x), ceil((float)height / threadsPerBlock.y));
	triangularSmooth_kernel<<<blocksPerGrid, threadsPerBlock>>>(grayImage_device, smoothImage_device, width, height, pitch);
	checkError(hipGetLastError(), "Failed to launch triangularSmooth_kernel (error code %s)\n");
	hipDeviceSynchronize();
	kernelTime.stop();

	// Copy the smooth image from the device to the host
	copyFromDeviceTime.start();
	error = hipMemcpy2D(smoothImage, width, smoothImage_device, pitch, width * sizeof(uchar), height, hipMemcpyDeviceToHost);
	checkError(error, "Failed to copy smoothImage from device to host (error code %s)\n");
	copyFromDeviceTime.stop();

	// Free the device buffers
	hipFree(grayImage_device);
	hipFree(smoothImage_device);

	// Print the timers
	cout << fixed << setprecision(6);
	//cout << "triangularSmooth (allocation): \t\t" << allocationTime.getElapsed() << " seconds." << endl;
	//cout << "triangularSmooth (copyToDevice): \t" << copyToDeviceTime.getElapsed() << " seconds." << endl;
	cout << "triangularSmooth (kernel): \t\t" << kernelTime.getElapsed() << " seconds." << endl;
	//cout << "triangularSmooth (copyFromDevice): \t" << copyFromDeviceTime.getElapsed() << " seconds." << endl;
}

